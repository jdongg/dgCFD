#include "hip/hip_runtime.h"
#include "DGsolve.hpp"
#include "meshing.hpp"

class DGsolver;

__device__
void volumeFluxF(int N, double *Q, double *F);

__device__
void volumeFluxG(int N, double *Q, double *G);

__device__
void numericalFluxF(int nOrder1D, double *QLeft, double *QRight, double *Fh);

__device__
void numericalFluxG(int nOrder1D, double *QLeft, double *QRight, double *Gh);

__device__
double myMax(int N, double *x);

__device__
double myMin(int N, double *x);

__device__
double minmod5(double a, double b, double c, double d, double e);

__device__ 
double minmod3(double a, double b, double c);

__device__
int sgn(double val);


/* computeRHS computes the spatial operator associated with the discrete scheme:
 *
 * 	u^{n+1} = u^{n} + dt*L(u^{n}),
 *	L(u^{n}) = -(F(u^{n}),dphix)_{K} - (G(u^{n}),dphiy)_{K} + surface terms.
 */
__global__ void 
computeRHScuda(int Nx, int Ny, int nElems, int nOrder2D, int nOrder1D, int nLoc, int *mapB, double *Q1, double *Q2, 
					double *Q3, double *Q4, double *rhsQ1, double *rhsQ2, double *rhsQ3, double *rhsQ4,
				    double *phiVol, double *dphixVol, double *dphiyVol, double *w2d, double *w1d,
				    double *phiEdgeLeft, double *phiEdgeRight, double *phiEdgeBottom, double *phiEdgeTop) {

	// we compute the RHS contributions in parallel on each element;
	// each thread is assigned a different element to work on. with OpenMP
	// shared memory, the variables xa, xb, ya, yb, vol, and the local
	// arrays to store neighbor coefficients and solution values must 
	// be intialized with the scope of the main for loop over nElems, otherwise
	// we will have a race condition. 
	int gtid = threadIdx.x + blockIdx.x*blockDim.x;
	for (int k = gtid; k < nElems; k += gridDim.x*blockDim.x) {
		// grab coordinates of vertices on element k
		double xa, xb, ya, yb, vol;

		// xa = Vxy[EToV[k*6]*2];
		// xb = Vxy[EToV[k*6+1]*2];
		// ya = Vxy[EToV[k*6]*2+1];
		// yb = Vxy[EToV[k*6+2]*2+1];

		int idx = k/Nx;
		int jdx = k%Nx;

		double x0 = 0.0;
		double y0 = 0.0;
		double xN = 0.25;
		double yN = 1.0;
		double hx = (xN-x0)/Nx;
		double hy = (yN-y0)/Ny;

		xa = x0 + idx*hx;
		xb = x0 + (idx+1)*hx;
		ya = y0 + jdx*hy;
		yb = y0 + (jdx+1)*hy;

		// Jacobian of transformation from reference element to
		// element k
		vol = (xb-xa)*(yb-ya)/4.0;

		// hard code these arrays into stack memory for now
		double c[4*9];
		double Qloc[4*16];
		double F[4*16];
		double G[4*16];
		double Fh[4*3];
		double Gh[4*3];

		// extract coefficients on cell k
		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			c[i] = Q1[k*nLoc+i];
			c[nLoc+i] = Q2[k*nLoc+i];
			c[2*nLoc+i] = Q3[k*nLoc+i];
			c[3*nLoc+i] = Q4[k*nLoc+i];
		}

		// compute conserved variables on element k
		// #pragma unroll
		for (int i = 0; i < nOrder2D; ++i)
		{
			Qloc[i] = 0.0;
			Qloc[nOrder2D+i] = 0.0;
			Qloc[2*nOrder2D+i] = 0.0;
			Qloc[3*nOrder2D+i] = 0.0;

			for (int j = 0; j < nLoc; ++j)
			{
				Qloc[i] += c[j]*phiVol[j*nOrder2D+i];
				Qloc[nOrder2D+i] += c[nLoc+j]*phiVol[j*nOrder2D+i];
				Qloc[2*nOrder2D+i] += c[2*nLoc+j]*phiVol[j*nOrder2D+i];
				Qloc[3*nOrder2D+i] += c[3*nLoc+j]*phiVol[j*nOrder2D+i];
			}
		}

		// compute volume fluxes on element k
		volumeFluxF(nOrder2D, Qloc, F);
		volumeFluxG(nOrder2D, Qloc, G);

		double rhsQ1tmp[9];
		double rhsQ2tmp[9];
		double rhsQ3tmp[9];
		double rhsQ4tmp[9];

		// compute volume contributions on element k
		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			// int idx = k*nLoc+i;

			rhsQ1tmp[i] = 0.0;
			rhsQ2tmp[i] = 0.0;
			rhsQ3tmp[i] = 0.0;
			rhsQ4tmp[i] = 0.0;

			for (int j = 0; j < nOrder2D; ++j)
			{
				int jdx = i*nOrder2D+j;
				rhsQ1tmp[i] += vol*w2d[j]*(dphixVol[jdx]*F[j] + dphiyVol[jdx]*G[j]);
				rhsQ2tmp[i] += vol*w2d[j]*(dphixVol[jdx]*F[nOrder2D+j] + dphiyVol[jdx]*G[nOrder2D+j]);
				rhsQ3tmp[i] += vol*w2d[j]*(dphixVol[jdx]*F[2*nOrder2D+j] + dphiyVol[jdx]*G[2*nOrder2D+j]);
				rhsQ4tmp[i] += vol*w2d[j]*(dphixVol[jdx]*F[3*nOrder2D+j] + dphiyVol[jdx]*G[3*nOrder2D+j]);

				// gravity source terms for Rayleigh-Taylor instability
				rhsQ3tmp[i] += vol*w2d[j]*Qloc[j]*phiVol[jdx];
				rhsQ4tmp[i] += vol*w2d[j]*Qloc[2*nOrder2D+j]*phiVol[jdx];
			}
		}


		// determine neighbor cells
		// int idx = EToV[k*6+4];
		// int jdx = EToV[k*6+5];

		// we should define these explicitly in order to avoid needing mapB
		// in unified memory
		int nLeft = mapB[k*4];
		int nRight = mapB[k*4+1];
		int nBottom = mapB[k*4+2];
		int nTop = mapB[k*4+3];

		// grab basis coefficients on neighbor cells
		double cLeft[9*4];
		double cRight[9*4];
		double cBottom[9*4];
		double cTop[9*4];

		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			cLeft[i] = Q1[nLeft*nLoc+i];
			cLeft[nLoc+i] = Q2[nLeft*nLoc+i];
			cLeft[2*nLoc+i] = Q3[nLeft*nLoc+i];
			cLeft[3*nLoc+i] = Q4[nLeft*nLoc+i];

			cRight[i] = Q1[nRight*nLoc+i];
			cRight[nLoc+i] = Q2[nRight*nLoc+i];
			cRight[2*nLoc+i] = Q3[nRight*nLoc+i];
			cRight[3*nLoc+i] = Q4[nRight*nLoc+i];

			cBottom[i] = Q1[nBottom*nLoc+i];
			cBottom[nLoc+i] = Q2[nBottom*nLoc+i];
			cBottom[2*nLoc+i] = Q3[nBottom*nLoc+i];
			cBottom[3*nLoc+i] = Q4[nBottom*nLoc+i];

			cTop[i] = Q1[nTop*nLoc+i];
			cTop[nLoc+i] = Q2[nTop*nLoc+i];
			cTop[2*nLoc+i] = Q3[nTop*nLoc+i];
			cTop[3*nLoc+i] = Q4[nTop*nLoc+i];
		}

		double Qedge[4*3];
		double Qneighbor[4*3];

		// left edge
		// #pragma unroll
		for (int i = 0; i < nOrder1D; ++i)
		{
			Qedge[i] = 0.0;
			Qedge[nOrder1D+i] = 0.0;
			Qedge[2*nOrder1D+i] = 0.0;
			Qedge[3*nOrder1D+i] = 0.0;

			Qneighbor[i] = 0.0;
			Qneighbor[nOrder1D+i] = 0.0;
			Qneighbor[2*nOrder1D+i] = 0.0;
			Qneighbor[3*nOrder1D+i] = 0.0;

			for (int j = 0; j < nLoc; ++j)
			{
				Qedge[i] += c[j]*phiEdgeLeft[j*nOrder1D+i];
				Qedge[nOrder1D+i] += c[nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
				Qedge[2*nOrder1D+i] += c[2*nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
				Qedge[3*nOrder1D+i] += c[3*nLoc+j]*phiEdgeLeft[j*nOrder1D+i];

				// if (jdx > 0)
				// {
					Qneighbor[i] += cLeft[j]*phiEdgeRight[j*nOrder1D+i];
					Qneighbor[nOrder1D+i] += cLeft[nLoc+j]*phiEdgeRight[j*nOrder1D+i];
					Qneighbor[2*nOrder1D+i] += cLeft[2*nLoc+j]*phiEdgeRight[j*nOrder1D+i];
					Qneighbor[3*nOrder1D+i] += cLeft[3*nLoc+j]*phiEdgeRight[j*nOrder1D+i];
				// }
				// else { // enforce natural BCs on the left boundary; left neighbor is itself
				// 	Qneighbor[i] += c[j]*phiEdgeLeft[j*nOrder1D+i];
				// 	Qneighbor[nOrder1D+i] += c[nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
				// 	Qneighbor[2*nOrder1D+i] += c[2*nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
				// 	Qneighbor[3*nOrder1D+i] += c[3*nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
				// }
			}
		}

		// compute numerical flux along left edge
		double vol1d = (xb-xa)/2.0;
		numericalFluxF(nOrder1D, Qedge, Qneighbor, Fh);

		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			// int idx = k*nLoc+i;
			for (int j = 0; j < nOrder1D; ++j)
			{
				rhsQ1tmp[i] += vol1d*w1d[j] * phiEdgeLeft[i*nOrder1D+j] * Fh[j];
				rhsQ2tmp[i] += vol1d*w1d[j] * phiEdgeLeft[i*nOrder1D+j] * Fh[nOrder1D+j];
				rhsQ3tmp[i] += vol1d*w1d[j] * phiEdgeLeft[i*nOrder1D+j] * Fh[2*nOrder1D+j];
				rhsQ4tmp[i] += vol1d*w1d[j] * phiEdgeLeft[i*nOrder1D+j] * Fh[3*nOrder1D+j];
			}
		}


		// bottom neighbor
		// #pragma unroll
		for (int i = 0; i < nOrder1D; ++i)
		{
			Qedge[i] = 0.0;
			Qedge[nOrder1D+i] = 0.0;
			Qedge[2*nOrder1D+i] = 0.0;
			Qedge[3*nOrder1D+i] = 0.0;

			Qneighbor[i] = 0.0;
			Qneighbor[nOrder1D+i] = 0.0;
			Qneighbor[2*nOrder1D+i] = 0.0;
			Qneighbor[3*nOrder1D+i] = 0.0;

			for (int j = 0; j < nLoc; ++j)
			{
				Qedge[i] += c[j]*phiEdgeBottom[j*nOrder1D+i];
				Qedge[nOrder1D+i] += c[nLoc+j]*phiEdgeBottom[j*nOrder1D+i];
				Qedge[2*nOrder1D+i] += c[2*nLoc+j]*phiEdgeBottom[j*nOrder1D+i];
				Qedge[3*nOrder1D+i] += c[3*nLoc+j]*phiEdgeBottom[j*nOrder1D+i];

				if (idx > 0)
				{
					Qneighbor[i] += cBottom[j]*phiEdgeTop[j*nOrder1D+i];
					Qneighbor[nOrder1D+i] += cBottom[nLoc+j]*phiEdgeTop[j*nOrder1D+i];
					Qneighbor[2*nOrder1D+i] += cBottom[2*nLoc+j]*phiEdgeTop[j*nOrder1D+i];
					Qneighbor[3*nOrder1D+i] += cBottom[3*nLoc+j]*phiEdgeTop[j*nOrder1D+i];
				}
				else { // enforce Dirichlet BCs on the bottom boundary
					Qneighbor[i] = 2.0;
					Qneighbor[nOrder1D+i] = 0.0;
					Qneighbor[2*nOrder1D+i] = 0.0;
					Qneighbor[3*nOrder1D+i] = 1.0/0.4;
				}
			}
		}

		// compute numerical flux along bottom boundary
		numericalFluxG(nOrder1D, Qedge, Qneighbor, Gh);

		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			// int idx = k*nLoc+i;
			for (int j = 0; j < nOrder1D; ++j)
			{
				rhsQ1tmp[i] += vol1d*w1d[j] * phiEdgeBottom[i*nOrder1D+j] * Gh[j];
				rhsQ2tmp[i] += vol1d*w1d[j] * phiEdgeBottom[i*nOrder1D+j] * Gh[nOrder1D+j];
				rhsQ3tmp[i] += vol1d*w1d[j] * phiEdgeBottom[i*nOrder1D+j] * Gh[2*nOrder1D+j];
				rhsQ4tmp[i] += vol1d*w1d[j] * phiEdgeBottom[i*nOrder1D+j] * Gh[3*nOrder1D+j];
			}
		}


		// right edge
		// #pragma unroll
		for (int i = 0; i < nOrder1D; ++i)
		{
			Qedge[i] = 0.0;
			Qedge[nOrder1D+i] = 0.0;
			Qedge[2*nOrder1D+i] = 0.0;
			Qedge[3*nOrder1D+i] = 0.0;

			Qneighbor[i] = 0.0;
			Qneighbor[nOrder1D+i] = 0.0;
			Qneighbor[2*nOrder1D+i] = 0.0;
			Qneighbor[3*nOrder1D+i] = 0.0;

			for (int j = 0; j < nLoc; ++j)
			{
				Qedge[i] += c[j]*phiEdgeRight[j*nOrder1D+i];
				Qedge[nOrder1D+i] += c[nLoc+j]*phiEdgeRight[j*nOrder1D+i];
				Qedge[2*nOrder1D+i] += c[2*nLoc+j]*phiEdgeRight[j*nOrder1D+i];
				Qedge[3*nOrder1D+i] += c[3*nLoc+j]*phiEdgeRight[j*nOrder1D+i];

				// if (jdx < Nx-1)
				// {
					Qneighbor[i] += cRight[j]*phiEdgeLeft[j*nOrder1D+i];
					Qneighbor[nOrder1D+i] += cRight[nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
					Qneighbor[2*nOrder1D+i] += cRight[2*nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
					Qneighbor[3*nOrder1D+i] += cRight[3*nLoc+j]*phiEdgeLeft[j*nOrder1D+i];
				// }
				// else { // enforce natural BCs along right edge; right neighbor is itself
				// 	Qneighbor[i] += c[j]*phiEdgeRight[j*nOrder1D+i];
				// 	Qneighbor[nOrder1D+i] += c[nLoc+j]*phiEdgeRight[j*nOrder1D+i];
				// 	Qneighbor[2*nOrder1D+i] += c[2*nLoc+j]*phiEdgeRight[j*nOrder1D+i];
				// 	Qneighbor[3*nOrder1D+i] += c[3*nLoc+j]*phiEdgeRight[j*nOrder1D+i];
				// }
			}
		}

		// compute numerical flux along right edge
		numericalFluxF(nOrder1D, Qneighbor, Qedge, Fh);

		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			// int idx = k*nLoc+i;
			for (int j = 0; j < nOrder1D; ++j)
			{
				rhsQ1tmp[i] -= vol1d*w1d[j] * phiEdgeRight[i*nOrder1D+j] * Fh[j];
				rhsQ2tmp[i] -= vol1d*w1d[j] * phiEdgeRight[i*nOrder1D+j] * Fh[nOrder1D+j];
				rhsQ3tmp[i] -= vol1d*w1d[j] * phiEdgeRight[i*nOrder1D+j] * Fh[2*nOrder1D+j];
				rhsQ4tmp[i] -= vol1d*w1d[j] * phiEdgeRight[i*nOrder1D+j] * Fh[3*nOrder1D+j];
			}
		}


		// top edge
		// #pragma unroll
		for (int i = 0; i < nOrder1D; ++i)
		{
			Qedge[i] = 0.0;
			Qedge[nOrder1D+i] = 0.0;
			Qedge[2*nOrder1D+i] = 0.0;
			Qedge[3*nOrder1D+i] = 0.0;

			Qneighbor[i] = 0.0;
			Qneighbor[nOrder1D+i] = 0.0;
			Qneighbor[2*nOrder1D+i] = 0.0;
			Qneighbor[3*nOrder1D+i] = 0.0;

			for (int j = 0; j < nLoc; ++j)
			{
				Qedge[i] += c[j]*phiEdgeTop[j*nOrder1D+i];
				Qedge[nOrder1D+i] += c[nLoc+j]*phiEdgeTop[j*nOrder1D+i];
				Qedge[2*nOrder1D+i] += c[2*nLoc+j]*phiEdgeTop[j*nOrder1D+i];
				Qedge[3*nOrder1D+i] += c[3*nLoc+j]*phiEdgeTop[j*nOrder1D+i];

				if (idx < Ny-1)
				{
					Qneighbor[i] += cTop[j]*phiEdgeBottom[j*nOrder1D+i];
					Qneighbor[nOrder1D+i] += cTop[nLoc+j]*phiEdgeBottom[j*nOrder1D+i];
					Qneighbor[2*nOrder1D+i] += cTop[2*nLoc+j]*phiEdgeBottom[j*nOrder1D+i];
					Qneighbor[3*nOrder1D+i] += cTop[3*nLoc+j]*phiEdgeBottom[j*nOrder1D+i];
				}
				else { // enforce Dirichlet BCs along top edge
					Qneighbor[i] = 1.0;
					Qneighbor[nOrder1D+i] = 0.0;
					Qneighbor[2*nOrder1D+i] = 0.0;
					Qneighbor[3*nOrder1D+i] = 2.5/0.4;
				}
			}
		}

		// compute numerical flux along top edge
		numericalFluxG(nOrder1D, Qneighbor, Qedge, Gh);

		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			// int idx = k*nLoc+i;
			for (int j = 0; j < nOrder1D; ++j)
			{
				rhsQ1tmp[i] -= vol1d*w1d[j] * phiEdgeTop[i*nOrder1D+j] * Gh[j];
				rhsQ2tmp[i] -= vol1d*w1d[j] * phiEdgeTop[i*nOrder1D+j] * Gh[nOrder1D+j];
				rhsQ3tmp[i] -= vol1d*w1d[j] * phiEdgeTop[i*nOrder1D+j] * Gh[2*nOrder1D+j];
				rhsQ4tmp[i] -= vol1d*w1d[j] * phiEdgeTop[i*nOrder1D+j] * Gh[3*nOrder1D+j];
			}
		}

		// we need to scale by the Jacobian because the fully-discrete scheme is
		//
		//		Mk*(u^{n+1}-u^{n}) = dt*L(u^{n}),
		//
		// where Mk is the local mass matrix. Since we are using the Legendre basis 
		// (i.e. an orthonormal basis), we have Mk = vol*Ik, where Ik is the identity
		// matrix. 
		// #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			int idx = k*nLoc+i;
			rhsQ1[idx] = rhsQ1tmp[i]/vol;
			rhsQ2[idx] = rhsQ2tmp[i]/vol;
			rhsQ3[idx] = rhsQ3tmp[i]/vol;
			rhsQ4[idx] = rhsQ4tmp[i]/vol;
		}
	}
}


__global__
void updateCUDA1(int nElems, int nLoc, double dt, double *Q1, double *Q2, double *Q3, double *Q4,
	             double *Q10, double *Q20, double *Q30, double *Q40, double *rhsQ1, double *rhsQ2,
	             double *rhsQ3, double *rhsQ4) {
	int gtid = threadIdx.x + blockIdx.x*blockDim.x;
	for (int i = gtid; i < nElems*nLoc; i += gridDim.x*blockDim.x)
	{
		Q1[i] = Q10[i] + dt*rhsQ1[i];
		Q2[i] = Q20[i] + dt*rhsQ2[i];
		Q3[i] = Q30[i] + dt*rhsQ3[i];
		Q4[i] = Q40[i] + dt*rhsQ4[i];
	}
}


__global__
void updateCUDA2(int nElems, int nLoc, double dt, double *Q1, double *Q2, double *Q3, double *Q4,
	             double *Q10, double *Q20, double *Q30, double *Q40, double *rhsQ1, double *rhsQ2,
	             double *rhsQ3, double *rhsQ4) {
	int gtid = threadIdx.x + blockIdx.x*blockDim.x;
	for (int i = gtid; i < nElems*nLoc; i += gridDim.x*blockDim.x)
	{
		Q1[i] = 3.0/4.0*Q10[i] + (Q1[i] + dt*rhsQ1[i])/4.0;
		Q2[i] = 3.0/4.0*Q20[i] + (Q2[i] + dt*rhsQ2[i])/4.0;
		Q3[i] = 3.0/4.0*Q30[i] + (Q3[i] + dt*rhsQ3[i])/4.0;
		Q4[i] = 3.0/4.0*Q40[i] + (Q4[i] + dt*rhsQ4[i])/4.0;
	}
}


__global__
void updateCUDA3(int nElems, int nLoc, double dt, double *Q1, double *Q2, double *Q3, double *Q4,
	             double *Q10, double *Q20, double *Q30, double *Q40, double *rhsQ1, double *rhsQ2,
	             double *rhsQ3, double *rhsQ4) {
	int gtid = threadIdx.x + blockIdx.x*blockDim.x;
	for (int i = gtid; i < nElems*nLoc; i += gridDim.x*blockDim.x)
	{
		Q10[i] = Q10[i]/3.0 + 2.0*(Q1[i] + dt*rhsQ1[i])/3.0;
		Q20[i] = Q20[i]/3.0 + 2.0*(Q2[i] + dt*rhsQ2[i])/3.0;
		Q30[i] = Q30[i]/3.0 + 2.0*(Q3[i] + dt*rhsQ3[i])/3.0;
		Q40[i] = Q40[i]/3.0 + 2.0*(Q4[i] + dt*rhsQ4[i])/3.0;
	}
}


__global__
void updateCUDA4(int nElems, int nLoc, double *Q1, double *Q2, double *Q3, double *Q4,
	             double *Q10, double *Q20, double *Q30, double *Q40) {
	int gtid = threadIdx.x + blockIdx.x*blockDim.x;
	for (int i = gtid; i < nElems*nLoc; i += gridDim.x*blockDim.x)
	{
		Q1[i] = Q10[i];
		Q2[i] = Q20[i];
		Q3[i] = Q30[i];
		Q4[i] = Q40[i];
	}
}


__device__
inline void volumeFluxF(int N, double *Q, double *F) {
	// #pragma unroll
	for (int i = 0; i < N; ++i)
	{
		double rho, v1, v2, p;

		rho = Q[i];
		v1 = Q[N+i]/rho;
		v2 = Q[2*N+i]/rho;
		p = (1.4-1.0)*(Q[3*N+i] - 0.5*rho*(v1*v1+v2*v2));

		F[i] = Q[N+i];
		F[N+i] = Q[N+i]*v1 + p;
		F[2*N+i] = Q[N+i]*v2;
		F[3*N+i] = v1*(Q[3*N+i] + p);
	}
}


__device__
inline void volumeFluxG(int N, double *Q, double *G) {
	// #pragma unroll
	for (int i = 0; i < N; ++i)
	{
		double rho, v1, v2, p;

		rho = Q[i];
		v1 = Q[N+i]/rho;
		v2 = Q[2*N+i]/rho;
		p = (1.4-1.0)*(Q[3*N+i] - 0.5*rho*(v1*v1+v2*v2));

		G[i] = Q[2*N+i];
		G[N+i] = Q[N+i]*v2;
		G[2*N+i] = Q[2*N+i]*v2 + p;
		G[3*N+i] = v2*(Q[3*N+i] + p);
	}
}


/* numericalFluxF computes the numerical flux of F along a given edge given
 * the two-sided values in QLeft and QRight. A local Lax-Friedrichs flux is used
 * here.
 *
 * \param [in] QLeft:  solution values of the left state
 * \param [in] QRight: solution values of the right state
 * \param [out] Fh:    4 x nOrder1D buffer to hold the numerical flux values
 */
__device__
inline void numericalFluxF(int nOrder1D, double *QLeft, double *QRight, double *Fh) {
	double fLeft[4*3];
	double fRight[4*3];
	double alphaF[4*3];

	volumeFluxF(nOrder1D, QLeft, fLeft);
	volumeFluxF(nOrder1D, QRight,fRight);

	// compute the maximum eigenvalue of F'(u) for Lax-Friedrichs flux
	// #pragma unroll
	for (int i = 0; i < nOrder1D; ++i)
	{
		double rho, v1, v2, p, c;

		rho = 0.5*(QLeft[i] + QRight[i]);
		v1 = 0.5*(QLeft[nOrder1D+i] + QRight[nOrder1D+i])/rho;
		v2 = 0.5*(QLeft[2*nOrder1D+i] + QRight[2*nOrder1D+i])/rho;
		p = 0.4*(0.5*(QLeft[3*nOrder1D+i] + QRight[3*nOrder1D+i]) - 0.5*rho*(v1*v1+v2*v2));
		c = sqrt(1.4*p/rho);

		alphaF[i] = fabs(v1-c);
		alphaF[nOrder1D+i] = fabs(v1);
		alphaF[2*nOrder1D+i] = fabs(v1);
		alphaF[3*nOrder1D+i] = fabs(v1+c);
	}

	// double lambda = 1.0;

	// TODO: should implement a naive max function inline here since
	// std functions will not work in CUDA kernels.
	// double lambda = *std::max_element(alphaF, alphaF+4*nOrder1D);
	double lambda = myMax(4*nOrder1D, alphaF);

	// #pragma unroll
	for (int i = 0; i < nOrder1D; ++i)
	{
		Fh[i] = 0.5*(fLeft[i] + fRight[i] - lambda*(QLeft[i] - QRight[i]));
		Fh[nOrder1D+i] = 0.5*(fLeft[nOrder1D+i] + fRight[nOrder1D+i] - lambda*(QLeft[nOrder1D+i] - QRight[nOrder1D+i]));
		Fh[2*nOrder1D+i] = 0.5*(fLeft[2*nOrder1D+i] + fRight[2*nOrder1D+i] - lambda*(QLeft[2*nOrder1D+i] - QRight[2*nOrder1D+i]));
		Fh[3*nOrder1D+i] = 0.5*(fLeft[3*nOrder1D+i] + fRight[3*nOrder1D+i] - lambda*(QLeft[3*nOrder1D+i] - QRight[3*nOrder1D+i]));
	}
}


/* numericalFluxG computes the numerical flux of G along a given edge given
 * the two-sided values in QLeft and QRight. A local Lax-Friedrichs flux is used
 * here.
 *
 * \param [in] QLeft:  solution values of the left state
 * \param [in] QRight: solution values of the right state
 * \param [out] Gh:    4 x nOrder1D buffer to hold the numerical flux values
 */
__device__
inline void numericalFluxG(int nOrder1D, double *QLeft, double *QRight, double *Gh) {
	double gLeft[4*3];
	double gRight[4*3];
	double alphaG[4*3];

	volumeFluxG(nOrder1D, QLeft, gLeft);
	volumeFluxG(nOrder1D, QRight,gRight);

	// compute maximum eigenvalue of G'(u)
	// #pragma unroll
	for (int i = 0; i < nOrder1D; ++i)
	{
		double rho, v1, v2, p, c;

		rho = 0.5*(QLeft[i] + QRight[i]);
		v1 = 0.5*(QLeft[nOrder1D+i] + QRight[nOrder1D+i])/rho;
		v2 = 0.5*(QLeft[2*nOrder1D+i] + QRight[2*nOrder1D+i])/rho;
		p = 0.4*(0.5*(QLeft[3*nOrder1D+i] + QRight[3*nOrder1D+i]) - 0.5*rho*(v1*v1+v2*v2));
		c = sqrt(1.4*p/rho);

		alphaG[i] = fabs(v2-c);
		alphaG[nOrder1D+i] = fabs(v2);
		alphaG[2*nOrder1D+i] = fabs(v2);
		alphaG[3*nOrder1D+i] = fabs(v2+c);
	}

	// double lambda = 1.0;
	// TODO: should implement a naive max function inline here since
	// std functions will not work in CUDA kernels.
	double lambda = myMax(4*nOrder1D, alphaG);

	// #pragma unroll
	for (int i = 0; i < nOrder1D; ++i)
	{
		Gh[i] = 0.5*(gLeft[i] + gRight[i] - lambda*(QLeft[i] - QRight[i]));
		Gh[nOrder1D+i] = 0.5*(gLeft[nOrder1D+i] + gRight[nOrder1D+i] - lambda*(QLeft[nOrder1D+i] - QRight[nOrder1D+i]));
		Gh[2*nOrder1D+i] = 0.5*(gLeft[2*nOrder1D+i] + gRight[2*nOrder1D+i] - lambda*(QLeft[2*nOrder1D+i] - QRight[2*nOrder1D+i]));
		Gh[3*nOrder1D+i] = 0.5*(gLeft[3*nOrder1D+i] + gRight[3*nOrder1D+i] - lambda*(QLeft[3*nOrder1D+i] - QRight[3*nOrder1D+i]));
	}
}


__device__
inline double myMax(int N, double *x) {
	double the_max = -10000.0;
	for (int i = 0; i < N; ++i)
	{
		if (x[i] > the_max)
		{
			the_max = x[i];
		}
	}

	return the_max;
}


__device__
inline double myMin(int N, double *x) {
	double the_min = 10000.0;
	for (int i = 0; i < N; ++i)
	{
		if (x[i] < the_min)
		{
			the_min = x[i];
		}
	}

	return the_min;
}


/* momentLimiter implements the moment-based slope limiter in 
 * https://www.sciencedirect.com/science/article/pii/S0021999107002136.
 * The main idea is to look at finite differences of "nearby" coefficients --
 * that is, the coefficients of the next lowest order basis functions -- and
 * use a typical minmod limiting approach. If the sign of the current coefficient
 * and the differences of neighboring coefficients are different, the coefficient
 * is set to zero and the basis function is effectively turned off.
 *
 * The moment limiter is an adaptive approach in that it first checks the highest
 * order coefficient(s). If it requires limiting, then we also check the next highest
 * order coefficients, otherwise we stop.
 *
 * The author notes that limiting of systems of conservation laws should be be
 * on the *characteristic* variables, not conserved variables. It may be a good idea
 * to considering adding this feature in the future. 
 *
 * \param[in/out] Q: nElems x nLoc buffer containing the conserved variables to
 *					 to be limited. The modified values are returned.
 */ 
__global__
void momentLimiterCUDA(int nElems, int nLoc, int pdeg, int *mapB, double *Q) {

	double tol = 1.e-12;

	int gtid = threadIdx.x + blockIdx.x*blockDim.x;
	for (int k = gtid; k < nElems; k += gridDim.x*blockDim.x)
	{
		// pull coefficients from current element
		double c[9];

		#pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			c[i] = Q[k*nLoc+i];
		}

		// lookup neighbor cells and pull coefficients
		int nLeft = mapB[k*4];
		int nRight = mapB[k*4+1];
		int nBottom = mapB[k*4+2];
		int nTop = mapB[k*4+3];

		double cLeft[9];
		double cRight[9];
		double cBottom[9];
		double cTop[9];
		double cmod[9];

		// parameters of the moment limiter; smaller values of ai and aj
		// lead to more diffusive solutions; larger values lead to less
		// limiting at the cost of potentially more oscillations

		// double ai = 0.75/sqrt(4.0*pdeg*pdeg-1.0);
		// double aj = 0.75/sqrt(4.0*pdeg*pdeg-1.0);

		double ai = 0.75*sqrt((2.0*pdeg-1.0)/(2.0*pdeg+1.0));
	    double aj = 0.75*sqrt((2.0*pdeg-1.0)/(2.0*pdeg+1.0));

	    #pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			cLeft[i] = Q[nLeft*nLoc+i];
			cRight[i] = Q[nRight*nLoc+i];
			cBottom[i] = Q[nBottom*nLoc+i];
			cTop[i] = Q[nTop*nLoc+i];

			cmod[i] = c[i];
		}

		// perform the limiting
		if (pdeg == 1)
		{
			cmod[3] = minmod5(c[3], aj*(cTop[2]-c[2]), aj*(c[2]-cBottom[2]),
				                    ai*(cRight[1]-c[1]), ai*(c[1]-cLeft[1]));

			if (fabs(cmod[3] - c[3]) > tol)
			{
				cmod[2] = minmod3(c[2], aj*(cRight[0]-c[0]), aj*(c[0]-cLeft[0]));
				cmod[1] = minmod3(c[1], ai*(cTop[0]-c[0]), ai*(c[0]-cBottom[0]));
			} else {
				cmod[2] = c[2];
				cmod[1] = c[1];
			}
		}
		else if (pdeg == 2)
		{
			cmod[8] = minmod5(c[8], aj*(cTop[7]-c[7]), aj*(c[7]-cBottom[7]),
									ai*(cRight[5]-c[5]), ai*(c[5]-cLeft[5]));

			if (fabs(cmod[8]-c[8]) > tol) 
			{
				cmod[7] = minmod5(c[7], aj*(cTop[6]-c[6]), aj*(c[6]-cBottom[6]),
									    ai*(cRight[4]-c[4]), ai*(c[4]-cLeft[4]));
				cmod[5] = minmod5(c[5], aj*(cTop[4]-c[4]), aj*(c[4] - cBottom[4]),
										ai*(cRight[2]-c[2]), ai*(c[2]-cLeft[2]));

				if (fabs(cmod[7]-c[7]) > tol && fabs(cmod[5]-c[5]) > tol)
				{
					cmod[2] = minmod3(cmod[2], aj*(cTop[1]-c[1]), aj*(c[1]-cBottom[1]));
					cmod[6] = minmod3(cmod[3], aj*(cRight[3]-c[3]), aj*(c[3]-cLeft[3]));

					if (fabs(cmod[2]-c[2]) > tol && fabs(cmod[6]-c[6]) > tol)
					{
						cmod[4] = minmod5(c[4], aj*(cTop[3]-c[3]), aj*(c[3]-cBottom[3]),
												ai*(cRight[1]-c[1]), aj*(c[1]-cLeft[1]));

						if (fabs(cmod[4]-c[4]) > tol)
						{
							cmod[1] = minmod3(c[1], aj*(cTop[0]-c[0]), aj*(c[0]-cBottom[0]));
							cmod[3] = minmod3(c[3], ai*(cRight[0]-c[0]), ai*(c[0]-cLeft[0]));
						}
					}
				}
			}
		}

		
		#pragma unroll
		for (int i = 0; i < nLoc; ++i)
		{
			Q[k*nLoc+i] = cmod[i];
		}
	}
}


/* minmod5 computes the minmod value of the five arguments specified. We have
 *
 * minmod(a,b,c,d,e) = sgn(a)*min(|a|,|b|,|c|,|d|,|e|)  if all arguments have same sign
 * 					 = 0								otherwise.
 *
 * \params [in] a,b,c,d,e: arguments of the minmod return
 * \param [out]:		   the minmod value
 */
__device__
inline double minmod5(double a, double b, double c, double d, double e) {
	if ((sgn(a)==sgn(b)) && (sgn(b)==sgn(c)) && (sgn(c)==sgn(d)) && (sgn(d)==sgn(e)) && (sgn(e)==sgn(a))) 
	{
		double array[5] = {fabs(a), fabs(b), fabs(c), fabs(d), fabs(e)};
		return sgn(a)*myMin(5, array);
	}
	else {
		return 0.0;
	}
}


/* minmod3 computes the minmod value of the three arguments specified. We have
 *
 * minmod(a,b,c) = sgn(a)*min(|a|,|b|,|c|)  if all arguments have same sign
 * 			     = 0					    otherwise.
 *
 * \params [in] a,b,c: arguments of the minmod return
 * \param [out]:	   the minmod value
 */
__device__
inline double minmod3(double a, double b, double c) {
	if ((sgn(a)==sgn(b)) && (sgn(b)==sgn(c)) && (sgn(c)==sgn(a)))
	{
		double array[3] = {fabs(a), fabs(b), fabs(c)};
		return sgn(a)*myMin(3, array);
	}
	else {
		return 0.0;
	}
}


/* sgn computes the signum function of the input
 * 
 * \param [in] val: argument of signum function
 * \param [out]:	signum(val)
 */
__device__
inline int sgn(double val) {
    // return (T(0) < val) - (val < T(0));
	if (val >= 0) return 1;
	if (val < 0) return -1;
	return 0;
}